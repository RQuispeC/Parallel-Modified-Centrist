
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#define TILE_WIDTH 32

#define COMMENT "Centrist_GPU"
#define RGB_COMPONENT_COLOR 255

typedef struct {
    unsigned char red, green, blue;
} PPMPixel;

typedef struct {
    int x, y;
    PPMPixel *data;
} PPMImage;

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}


static PPMImage *readPPM(const char *filename) {
    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;
    fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    if (!fgets(buff, sizeof(buff), fp)) {
        perror(filename);
        exit(1);
    }

    if (buff[0] != 'P' || buff[1] != '6') {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    img = (PPMImage *) malloc(sizeof(PPMImage));
    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    c = getc(fp);
    while (c == '#') {
        while (getc(fp) != '\n')
            ;
        c = getc(fp);
    }

    ungetc(c, fp);
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
        fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
        exit(1);
    }

    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
        fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
                filename);
        exit(1);
    }

    if (rgb_comp_color != RGB_COMPONENT_COLOR) {
        fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
        exit(1);
    }

    while (fgetc(fp) != '\n')
        ;
    img->data = (PPMPixel*) malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
        fprintf(stderr, "Error loading image '%s'\n", filename);
        exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(PPMImage *img) {

    fprintf(stdout, "P6\n");
    fprintf(stdout, "# %s\n", COMMENT);
    fprintf(stdout, "%d %d\n", img->x, img->y);
    fprintf(stdout, "%d\n", RGB_COMPONENT_COLOR);

    fwrite(img->data, 3 * img->x, img->y, stdout);
    fclose(stdout);
}
//create a thread per pixel
__global__ void mod_CENTRIST(PPMPixel *image_out, PPMPixel *image_cp, int columns, int rows, int *hist, int hist_len) {

	int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
	int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

  __shared__ int hist_private[512];
  int hist_index = (threadIdx.y*TILE_WIDTH + threadIdx.x); //get index in shared histogram
  if(hist_index < hist_len) hist_private[hist_index] = 0;
  __syncthreads();
	if(col < columns && row < rows)
	{
    //create and copy small chunks to shared memory
    __shared__ unsigned char image_cp_private[TILE_WIDTH][TILE_WIDTH];

    //convert to grayscale
		int img_index = row * columns + col; //get index in original image
    int grayscale = (image_cp[img_index].red*299 + image_cp[img_index].green*587 + image_cp[img_index].blue*114)/1000; //avoid float point errors

    image_cp_private[threadIdx.y][threadIdx.x] = grayscale;

    __syncthreads();
    if(col < columns - 2 && row < rows - 2) //ignore first/last row/column
    {
      int r, c, rr, cc;
      float mean = 0.0;
      for(r = threadIdx.y, rr = row; r <= threadIdx.y + 2; r++, rr++)
        for(c = threadIdx.x , cc = col; c <= threadIdx.x + 2; c++, cc++)
        {
          if(r < TILE_WIDTH && c < TILE_WIDTH)
          {
            mean += image_cp_private[r][c];
          }
          else
          {
            int grayscale_neigh = (image_cp[rr*columns + cc].red*299 + image_cp[rr*columns + cc].green*587 + image_cp[rr*columns + cc].blue*114)/1000;
            mean += grayscale_neigh;
          }
        }
      mean /= 9.0;
      int value = 0, k = 8;
      for(r = threadIdx.y, rr = row ; r <= threadIdx.y + 2; r++, rr++)
        for(c = threadIdx.x, cc = col ; c <= threadIdx.x + 2; c++, cc++)
        {
          if(r < TILE_WIDTH && c < TILE_WIDTH)
          {
            if(1.0*image_cp_private[r][c] >= mean)
              value |= 1<<k;
          }
          else
          {
            int grayscale_neigh = (image_cp[rr*columns + cc].red*299 + image_cp[rr*columns + cc].green*587 + image_cp[rr*columns + cc].blue*114)/1000;
            if(grayscale_neigh >= mean)
              value |= 1<<k;
          }
          k--;
        }
      int img_out_ind = row * (columns - 2) + col; //get index in ouput original
      image_out[img_out_ind].red = image_out[img_out_ind].blue = image_out[img_out_ind].green = value;
      atomicAdd(&(hist_private[value]), 1);
    }
    __syncthreads();
    if(hist_index == 0)
    {
      for(int i = 0; i < hist_len; i++)
        atomicAdd(&(hist[i]), hist_private[i]); //init shared histogram
    }
	}
}

int main(int argc, char *argv[]) {

    if( argc != 2 ) {
        printf("Too many or no one arguments supplied.\n");
    }

    double t_start, t_end;
    char *filename = argv[1]; //Recebendo o arquivo!;

    PPMImage *image = readPPM(filename);
    PPMImage *image_output = readPPM(filename);
    int *hist;

    //device data
  	PPMPixel *d_image_output;
    PPMPixel *d_image_copy;
    int *d_hist;

    //total excecution time
  	double offload=0.0 , kernel = 0.0;

  	int i_size = sizeof(PPMPixel) * image->x * image->y;
    int hist_len = 512;
    int hist_size = sizeof(int)*hist_len;

    hist =(int *)malloc(hist_size);
    int i;
    for(i = 0; i < hist_len; i++) hist[i] = 0;

  	// Allocate space for device copies of image and h
  	t_start = rtclock();
  	hipMalloc(&d_image_output, i_size);
  	hipMalloc(&d_image_copy, i_size);
    hipMalloc(&d_hist, hist_size);
  	t_end = rtclock();

    fprintf(stdout, "CudaMalloc %0.6lfs\n", t_end - t_start);
  	offload +=  t_end - t_start;

    //copy inputs to device
  	t_start = rtclock();
  	hipMemcpy(d_image_output, image_output->data, i_size, hipMemcpyHostToDevice);
    hipMemcpy(d_image_copy, image->data, i_size, hipMemcpyHostToDevice);
    hipMemcpy(d_hist, hist, hist_size, hipMemcpyHostToDevice);
  	t_end = rtclock();

  	fprintf(stdout, "CopyToDevice  %0.6lfs\n", t_end - t_start);
  	offload +=  t_end - t_start;

    //set grids size
  	dim3 dimGrid(ceil((float)image -> x / TILE_WIDTH), ceil((float)image -> y / TILE_WIDTH), 1);
  	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  	t_start = rtclock();
  	mod_CENTRIST<<<dimGrid, dimBlock>>>(d_image_output, d_image_copy, image-> x, image -> y, d_hist, hist_len);
  	hipDeviceSynchronize();
  	t_end = rtclock();

  	fprintf(stdout, "Kernel  %0.6lfs\n", t_end - t_start);
  	kernel +=  t_end - t_start;

  	t_start = rtclock();
  	hipMemcpy(image_output->data, d_image_output, i_size, hipMemcpyDeviceToHost);
    hipMemcpy(hist, d_hist, hist_size, hipMemcpyDeviceToHost);
  	t_end = rtclock();

  	fprintf(stdout, "CopyFromDevice %0.6lfs\n", t_end - t_start);
  	offload +=  t_end - t_start;

    fprintf(stdout, "Offload %0.6lfs\n", offload);
    fprintf(stdout, "CudaTotal %0.6lfs\n", offload + kernel);

    image_output->x -= 2;
    image_output->y -= 2;
    float dim = (float)((image_output->x) * (image_output->y));

    //writePPM(image_output);
    //for(i=0; i < hist_len; i++) printf("%.4f ", hist[i]/dim);

    free(image);
    free(image_output);
    hipFree(d_image_output), hipFree(d_image_copy), hipFree(d_hist);
}


